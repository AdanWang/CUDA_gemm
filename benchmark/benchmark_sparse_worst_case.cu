#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "bcsr.hpp"
#include "csr.hpp"
#include "utils.hpp"
#include "sparse_help_func.hpp"
#include "sparse.cu"


int main(int argc, char** argv) {
    if (argc != 5) {
        printf("usage: ./main [M] [K] [N] [Sparsity]\n");
        exit(0);
    }
    size_t M = atoi(argv[1]);
    size_t K = atoi(argv[2]);
    size_t N = atoi(argv[3]);
    size_t Sparsity = atoi(argv[4]);

    size_t bytes = sizeof(float) * M * K;
    float* h_A = (float*)malloc(bytes);
    float* h_B = (float*)malloc(bytes);
    float* h_C = (float*)malloc(bytes);
    float* h_C1 = (float*)malloc(bytes);

    float* d_A;
    float* d_B;
    float* d_C;

    checkCudaErrors(hipMalloc(&d_A, bytes));
    checkCudaErrors(hipMalloc(&d_B, bytes));
    checkCudaErrors(hipMalloc(&d_C, bytes));
    double msecPerMatrixMul[2] = {0, 0};
    double gigaFlops[2] = {0, 0};
    double flopsPerMatrixMul = 2.0 * M * N * K;

    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_K = 32;
    const int BLOCK_SIZE_N = 32;
    const int THREAD_SIZE_X = 4;
    const int THREAD_SIZE_Y = 4;
    const bool ENABLE_DOUBLE_BUFFER = false;

    float alpha = 1.0;
    float beta = 0;
    
    // 生成A的数据
    // worst case random 
    int nnz = M * K * (Sparsity / 100.0);
    int nnz_stride = M * K / nnz;
    for ( int i = 0; i < M * K; i++ ) {
            if (i % nnz_stride == 0) h_A[i] = 1;
            else {
                h_A[i] = 0;
            }
        }

    // 生成B的数据
    for( int i = 0 ; i < K; i ++ ) {
        for ( int j = 0 ; j < N ; j ++) {
            if ( i < K / 2 && j < N / 2) h_B[i * N + j] = 0;
            else if ( i < K / 2 && j >= N / 2) h_B[i * N + j] = 1;
            else if ( i >= K / 2 && j < N / 2) h_B[i * N + j] = 2;
            else {
                h_B[i * N + j] = 3;
            }
        }
    }
    
    checkCudaErrors(hipMemcpy( d_A, h_A, bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_B, h_B, bytes, hipMemcpyHostToDevice));
    
    
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float msecTotal = 0;
    int nIter = 100;

    // bcsr
    // convert to bcsr mat
    bcsr bcsr_mat{(int)M, (int)K, BLOCK_SIZE_M, BLOCK_SIZE_K};
    cal_block(&bcsr_mat, h_A);

    bcsr_mat.row_ptr = (int*)malloc(sizeof(int) * ( bcsr_mat.m_block + 1 ));
    bcsr_mat.col_idx = (int*)malloc(sizeof(int) * bcsr_mat.nnz_block_num );
    bcsr_mat.val = (float*)malloc(sizeof(float) * bcsr_mat.nnz_block_num * bcsr_mat.m_block_sz * bcsr_mat.n_block_sz);
    
    generate_bcsr(&bcsr_mat, h_A);


    float* val;
    int* col_idx;
    int* row_ptr;

    checkCudaErrors(hipMalloc(&val, sizeof(float) * bcsr_mat.nnz_block_num * bcsr_mat.m_block_sz * bcsr_mat.n_block_sz));
    checkCudaErrors(hipMalloc(&col_idx, sizeof(int) * bcsr_mat.nnz_block_num));
    checkCudaErrors(hipMalloc(&row_ptr, sizeof(int) * ( bcsr_mat.m_block + 1 )));
    
    checkCudaErrors(hipMemcpy( val, bcsr_mat.val, sizeof(float) * bcsr_mat.nnz_block_num * bcsr_mat.m_block_sz * bcsr_mat.n_block_sz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( col_idx, bcsr_mat.col_idx, sizeof(int) * bcsr_mat.nnz_block_num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( row_ptr, bcsr_mat.row_ptr, sizeof(int) * ( bcsr_mat.m_block + 1), hipMemcpyHostToDevice));
    
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
        dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
        MatrixMulCUDA5<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X, ENABLE_DOUBLE_BUFFER>
        <<< dimGrid, dimBlock >>>(val, col_idx, row_ptr, d_B, d_C, K, N);
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy( h_C, d_C, bytes, hipMemcpyDeviceToHost));

    msecPerMatrixMul[0] = msecTotal / nIter;
    gigaFlops[0] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf( "My sparse block gemm Performance= %.0f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[0],
        msecPerMatrixMul[0],
        flopsPerMatrixMul);

    // cusparse csr
    csr csr_mat{(int)M, (int)K}; 
    cal_nnz(&csr_mat, h_A);

    csr_mat.row_ptr = (int*)malloc(sizeof(int) * ( csr_mat.m + 1 ));
    csr_mat.col_idx = (int*)malloc(sizeof(int) * csr_mat.nnz_num );
    csr_mat.val = (float*)malloc(sizeof(float) * csr_mat.nnz_num );

    generate_csr(&csr_mat, h_A);
    
    float* csr_val;
    int* csr_col_idx;
    int* csr_row_ptr;

    checkCudaErrors(hipMalloc(&csr_val, sizeof(float) * csr_mat.nnz_num));
    checkCudaErrors(hipMalloc(&csr_col_idx, sizeof(int) * csr_mat.nnz_num));
    checkCudaErrors(hipMalloc(&csr_row_ptr, sizeof(int) * ( csr_mat.m + 1 )));
    
    checkCudaErrors(hipMemcpy( csr_val, csr_mat.val, sizeof(float) * csr_mat.nnz_num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( csr_col_idx, csr_mat.col_idx, sizeof(int) * csr_mat.nnz_num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( csr_row_ptr, csr_mat.row_ptr, sizeof(int) * ( csr_mat.m + 1 ), hipMemcpyHostToDevice));

    hipsparseHandle_t cusparse_handle;
    hipsparseSpMatDescr_t descrA;
    hipsparseDnMatDescr_t descrB, descrC;

    checkCuSparseErrors(hipsparseCreate(&cusparse_handle));

    checkCuSparseErrors(
        hipsparseCreateDnMat(
            &descrB,
            K,
            M,
            K,
            d_B,
            HIP_R_32F,
            HIPSPARSE_ORDER_COL));
    
    checkCuSparseErrors(
        hipsparseCreateDnMat(
            &descrC,
            M,
            N,
            M,
            d_C,
            HIP_R_32F,
            HIPSPARSE_ORDER_COL));
    
    checkCuSparseErrors (
        hipsparseCreateCsr(&descrA,
            M,
            K,
            csr_mat.nnz_num,
            csr_row_ptr,
            csr_col_idx,
            csr_val,
            HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO,
            HIP_R_32F));

    checkCudaErrors(hipMemcpy( d_B, h_B, bytes, hipMemcpyHostToDevice));
    
    size_t buffer_size;
    checkCuSparseErrors(
        hipsparseSpMM_bufferSize(
                cusparse_handle,
                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                HIPSPARSE_OPERATION_TRANSPOSE,
                &alpha,
                descrA,
                descrB,
                &beta,
                descrC,
                HIP_R_32F,
                HIPSPARSE_CSRMM_ALG1,
                &buffer_size
            ));
    float* externalBuffer;
    checkCudaErrors(hipMalloc(&externalBuffer, buffer_size));
    
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        checkCuSparseErrors(
            hipsparseSpMM(cusparse_handle,
                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                HIPSPARSE_OPERATION_TRANSPOSE,
                &alpha,
                descrA,
                descrB,
                &beta,
                descrC,
                HIP_R_32F,
                HIPSPARSE_CSRMM_ALG1,
                externalBuffer
                )
            );
        
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy( h_C1, d_C, bytes, hipMemcpyDeviceToHost));

    msecPerMatrixMul[1] = msecTotal / nIter;
    gigaFlops[1] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
    printf( "CuSparse Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[1],
        msecPerMatrixMul[1],
        flopsPerMatrixMul);
    
    double eps = 1.e-6;  // machine zero
    bool correct = true;
    for (int i = 0; i < M * N; i++) {
        // h_C1 是转置
        int row = i / N;
        int col = i % N;
        double abs_err = fabs(h_C[i] - h_C1[col * M + row]);
        double dot_length = M;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                    i, h_C[i], h_C1[col * M + row], eps);
            correct = false;
            break;
        }
    }



    printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
    printf("ratio= %f\n", gigaFlops[0] / gigaFlops[1]);
    
    // Free Memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C1);
}