#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "dense_help_func.hpp"
#include "dense.cu"
#include "encoding.cu"

int main(int argc, char** argv) {
    if (argc != 4) {
        printf("usage: ./main [M] [K] [N]\n");
        exit(0);
    }
    size_t M = atoi(argv[1]);
    size_t K = atoi(argv[2]);
    size_t N = atoi(argv[3]);

    size_t bytes = sizeof(float) * M * K;
    float* h_A = (float*)malloc(bytes);
    float* h_B = (float*)malloc(bytes);
    float* h_C = (float*)malloc(bytes);
    float* h_C1 = (float*)malloc(bytes);

    float* d_A;
    float* d_B;
    float* d_C;

    checkCudaErrors(hipMalloc(&d_A, bytes));
    checkCudaErrors(hipMalloc(&d_B, bytes));
    checkCudaErrors(hipMalloc(&d_C, bytes));
    double msecPerMatrixMul[2] = {0, 0};
    double gigaFlops[2] = {0, 0};
    double flopsPerMatrixMul = 2.0 * M * N * K;

    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_K = 32;
    const int BLOCK_SIZE_N = 32;
    const int THREAD_SIZE_X = 4;
    const int THREAD_SIZE_Y = 4;
    const bool ENABLE_DOUBLE_BUFFER = false;
    int k_block = K / BLOCK_SIZE_K;
    int stride = 2;

    // 生成A的数据
    for( int i = 0; i < M * K; i++ ) {
        int row = (i / K);
        int col = (i % K);
        int row_block = row / BLOCK_SIZE_M;
        int col_block = col / BLOCK_SIZE_K;
        if ((row_block * k_block + col_block) % stride == 0) h_A[i] = 1;
        else {
            h_A[i] = 0;
        }
    }

    // 生成B的数据
    for( int i = 0; i < K * N; i++ ) {
        if ( i >= K * N / 2) h_B[i] = 2;
        else {
            h_B[i] = 0;
        }
    }

    checkCudaErrors(hipMemcpy( d_A, h_A, bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_B, h_B, bytes, hipMemcpyHostToDevice));
    
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float msecTotal = 0;
    int nIter = 100;

    checkCudaErrors(hipMemcpy( d_C, h_C, bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));

    for (int run = 0 ; run < nIter; run ++ ) {
        dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
        dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
        MatrixMulCUDA6<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X, ENABLE_DOUBLE_BUFFER> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, K, N);

    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));


    checkCudaErrors(hipMemcpy( h_C, d_C, bytes, hipMemcpyDeviceToHost));

    msecPerMatrixMul[0] = msecTotal / nIter;
    gigaFlops[0] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf( "My gemm Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[0],
        msecPerMatrixMul[0],
        flopsPerMatrixMul);

    // encoding
    checkCudaErrors(hipMemcpy( d_C, h_C, bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
        dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
        MatrixMulCUDAEncoding<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X, ENABLE_DOUBLE_BUFFER> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, K, N);
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy( h_C1, d_C, bytes, hipMemcpyDeviceToHost));

    msecPerMatrixMul[1] = msecTotal / nIter;
    gigaFlops[1] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
    printf( "encoding Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[1],
        msecPerMatrixMul[1],
        flopsPerMatrixMul);

    printf("ratio= %f\n", gigaFlops[0] / gigaFlops[1]);
    
    // Free Memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C1);
}