#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "dense_help_func.hpp"
#include "quantization_8bit.cu"
#include "dense.cu"

int main(int argc, char** argv) {
    if (argc != 4) {
        printf("usage: ./main [M] [K] [N]\n");
        exit(0);
    }
    size_t M = atoi(argv[1]);
    size_t K = atoi(argv[2]);
    size_t N = atoi(argv[3]);

    // for uint8
    size_t bytes = sizeof(uint32_t) * M * K / 4;
    uint32_t* h_A = (uint32_t*)malloc(bytes);
    uint32_t* h_B = (uint32_t*)malloc(bytes);
    uint32_t* h_C = (uint32_t*)malloc(bytes);

    uint32_t* d_A;
    uint32_t* d_B;
    uint32_t* d_C;

    checkCudaErrors(hipMalloc(&d_A, bytes));
    checkCudaErrors(hipMalloc(&d_B, bytes));
    checkCudaErrors(hipMalloc(&d_C, bytes));

    // for float
    size_t fbytes = sizeof(float) * M * K;
    float* fh_A = (float*)malloc(fbytes);
    float* fh_B = (float*)malloc(fbytes);
    float* fh_C = (float*)malloc(fbytes);

    float* fd_A;
    float* fd_B;
    float* fd_C;

    checkCudaErrors(hipMalloc(&fd_A, fbytes));
    checkCudaErrors(hipMalloc(&fd_B, fbytes));
    checkCudaErrors(hipMalloc(&fd_C, fbytes));

    double msecPerMatrixMul[2] = {0, 0};
    double gigaFlops[2] = {0, 0};
    double flopsPerMatrixMul = 2.0 * M * N * K;

    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_K = 32;
    const int BLOCK_SIZE_N = 32;
    const int THREAD_SIZE_X = 4;
    const int THREAD_SIZE_Y = 4;
    const bool ENABLE_DOUBLE_BUFFER = false;
    const int BIT_WIDTH = 8;
    int k_block = K / BLOCK_SIZE_K;
    int stride = 2;

    // 生成A的数据
    for( int i = 0; i < M * K; i++ ) {
        int row = (i / K);
        int col = (i % K);
        int row_block = row / BLOCK_SIZE_M;
        int col_block = col / BLOCK_SIZE_K;
        if ((row_block * k_block + col_block) % stride == 0) {
            h_A[i/4] = 0x01010101;
            fh_A[i] = 1;
        }
        else {
            h_A[i/4] = 0;
            fh_A[i] = 0;
        }
    }

    // 生成B的数据
    for( int i = 0; i < K * N; i++ ) {
        if ( i >= K * N / 2) {
            h_B[i/4] = 0x02020202;
            fh_B[i] = 2;
        }
        else {
            h_B[i/4] = 0;
            fh_B[i] = 0;
        }
    }

    checkCudaErrors(hipMemcpy( d_A, h_A, bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_B, h_B, bytes, hipMemcpyHostToDevice));
    
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float msecTotal = 0;
    int nIter = 100;

    checkCudaErrors(hipMemcpy( d_C, h_C, bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));

    for (int run = 0 ; run < nIter; run ++ ) {
        dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
        dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
        MatrixMulCUDAQuantize8bit<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X, BIT_WIDTH, ENABLE_DOUBLE_BUFFER> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, K, N);

    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));


    checkCudaErrors(hipMemcpy( h_C, d_C, bytes, hipMemcpyDeviceToHost));

    msecPerMatrixMul[0] = msecTotal / nIter;
    gigaFlops[0] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf( "My gemm Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[0],
        msecPerMatrixMul[0],
        flopsPerMatrixMul);

    // dense
    checkCudaErrors(hipMemcpy( fd_A, fh_A, fbytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( fd_B, fh_B, fbytes, hipMemcpyHostToDevice));
    

    checkCudaErrors(hipMemcpy( fd_C, fh_C, fbytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));

    for (int run = 0 ; run < nIter; run ++ ) {
        dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
        dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
        MatrixMulCUDA6<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X, ENABLE_DOUBLE_BUFFER> 
        <<< dimGrid, dimBlock >>>(fd_A, fd_B, fd_C, K, N);

    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));


    checkCudaErrors(hipMemcpy( fh_C, fd_C, fbytes, hipMemcpyDeviceToHost));

    msecPerMatrixMul[1] = msecTotal / nIter;
    gigaFlops[1] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
    printf( "My gemm1 Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[1],
        msecPerMatrixMul[1],
        flopsPerMatrixMul);

    
    double eps = 1.e-6;  // machine zero
    bool correct = true;
    for (int i = 0; i < M * N; i++) {
        int row = i / N;
        int col = i % N;
        double abs_err = fabs(h_C[i/4] - fh_C[col * M + row]);
        double dot_length = M;
        double abs_val = fabs(h_C[i/4]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                    i, (float)h_C[i], fh_C[col * M + row], eps);
            correct = false;
            break;
        }
    }

    printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
    printf("ratio= %f\n", gigaFlops[0] / gigaFlops[1]);
    
    // Free Memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    free(h_A);
    free(h_B);
    free(h_C);


    hipFree(fd_A);
    hipFree(fd_B);
    hipFree(fd_C);
    free(fh_A);
    free(fh_B);
    free(fh_C);
}