#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

// Macro to catch CUDA errors in CUDA runtime calls
#define CUDA_SAFE_CALL(call)                                          \
do {                                                                  \
    hipError_t err = call;                                           \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err) );       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)
// Macro to catch CUDA errors in kernel launches
#define CHECK_LAUNCH_ERROR()                                          \
do {                                                                  \
    /* Check synchronous errors, i.e. pre-launch */                   \
    hipError_t err = hipGetLastError();                             \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err) );       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
    /* Check asynchronous errors, i.e. kernel failed (ULF) */         \
    err = hipDeviceSynchronize();                                    \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString( err) );      \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)

texture<float, 2, hipReadModeElementType> tex;

__global__ void kernel (int m, int n, float* a) 
{
    float val;
    int i = 0;
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            val = tex2D (tex, col, row);
            printf ("%f  %f\n", val, a[i++]);
        }
        printf ("\n");
    }
}

int main (void)
{
    int m = 4; // height = #rows
    int n = 3; // width  = #columns
    size_t pitch, tex_ofs;
    float arr[4][3]= {{10.1, 11, 12},
                    {20, 21, 22},
                    {30, 31, 32},
                    {40, 41, 42}};
    float *arr_d;

    CUDA_SAFE_CALL(hipMallocPitch((void**)&arr_d,&pitch,n*sizeof(*arr_d),m));
    CUDA_SAFE_CALL(hipMemcpy2D(arr_d, pitch, arr, n*sizeof(arr[0][0]), n*sizeof(arr[0][0]), m, hipMemcpyHostToDevice));
    tex.normalized = false;
    CUDA_SAFE_CALL (hipBindTexture2D (&tex_ofs, &tex, arr_d, &tex.channelDesc,
                                       n, m, pitch));
    if (tex_ofs !=0) {
        printf ("tex_ofs = %zu\n", tex_ofs);
        return EXIT_FAILURE;
    }
    printf ("reading texture:\n");
    kernel<<<1,1>>>(m, n, arr_d);
    CHECK_LAUNCH_ERROR();
    CUDA_SAFE_CALL (hipDeviceSynchronize());
    return EXIT_SUCCESS;
}