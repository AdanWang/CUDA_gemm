#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "bcsr.hpp"
#include "csr.hpp"
#include "utils.hpp"
#include "sparse_help_func.hpp"

// template <int BLOCK_SIZE> __global__ void MatrixMulCUDA5( 
//     float * __restrict__ A_Val,
//     int* __restrict__ A_col_idx,
//     int* __restrict__ A_row_ptr,
//     float * __restrict__ B,
//     float * __restrict__ C, 
//     const int K,
//     const int N) {
//     // Block index
//     int bx = blockIdx.x;
//     int by = blockIdx.y;

//     // Thread index
//     int tx = threadIdx.x;
//     int ty = threadIdx.y;
    
//     float4 Csub[4] = {
//         {0, 0, 0, 0},
//         {0, 0, 0, 0},
//         {0, 0, 0, 0},
//         {0, 0, 0, 0}};
    
//     int row_ptr_start = A_row_ptr[by];
//     int row_ptr_end = A_row_ptr[by + 1];
    
//     for (int row_ptr = row_ptr_start ; row_ptr < row_ptr_end ; row_ptr = row_ptr + 1) {
//         int tile_idx = A_col_idx[row_ptr];
//         __shared__ float As[BLOCK_SIZE * BLOCK_SIZE];
//         __shared__ float Bs[BLOCK_SIZE * BLOCK_SIZE];
//         float* A = A_Val + BLOCK_SIZE * BLOCK_SIZE * row_ptr;
//         #pragma unroll
//         for ( int i = 0 ; i < 4 ; i ++ ) {
//             reinterpret_cast<float4*>(As + BLOCK_SIZE * (ty * 4 + i) + tx * 4)[0] 
//                 = reinterpret_cast<float4*>( A + BLOCK_SIZE * (ty * 4 + i) + tx * 4 )[0];
            
//                 reinterpret_cast<float4*>(Bs + BLOCK_SIZE * (ty * 4 + i) + tx * 4)[0] 
//                 = reinterpret_cast<float4*>(B + (BLOCK_SIZE * tile_idx + ty * 4 + i ) * N + BLOCK_SIZE * bx + tx * 4 )[0];
//         }
    
//         __syncthreads();

//         #pragma unroll
//         for (int k = 0; k < BLOCK_SIZE; ++k) {
            
//             Csub[0].x = fma(As[ty * 4 * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4], Csub[0].x);
//             Csub[0].y = fma(As[ty * 4 * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 1], Csub[0].y);
//             Csub[0].z = fma(As[ty * 4 * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 2], Csub[0].z);
//             Csub[0].w = fma(As[ty * 4 * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 3], Csub[0].w);
//             Csub[1].x = fma(As[(ty * 4 + 1) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4], Csub[1].x);
//             Csub[1].y = fma(As[(ty * 4 + 1) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 1], Csub[1].y);
//             Csub[1].z = fma(As[(ty * 4 + 1) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 2], Csub[1].z);
//             Csub[1].w = fma(As[(ty * 4 + 1) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 3], Csub[1].w);
//             Csub[2].x = fma(As[(ty * 4 + 2) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4], Csub[2].x);
//             Csub[2].y = fma(As[(ty * 4 + 2) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 1], Csub[2].y);
//             Csub[2].z = fma(As[(ty * 4 + 2) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 2], Csub[2].z);
//             Csub[2].w = fma(As[(ty * 4 + 2) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 3], Csub[2].w);
//             Csub[3].x = fma(As[(ty * 4 + 3) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4], Csub[3].x);
//             Csub[3].y = fma(As[(ty * 4 + 3) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 1], Csub[3].y);
//             Csub[3].z = fma(As[(ty * 4 + 3) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 2], Csub[3].z);
//             Csub[3].w = fma(As[(ty * 4 + 3) * BLOCK_SIZE + k], Bs[k * BLOCK_SIZE + tx * 4 + 3], Csub[3].w);
            
//         }
//         // wait threads to finish , otherwise next tile will overwrite the shared memory
//         __syncthreads();
//     }

//     reinterpret_cast<float4*> (C + N * ( BLOCK_SIZE * by + ty * 4 ) + BLOCK_SIZE * bx + tx * 4 )[0] = Csub[0];
//     reinterpret_cast<float4*> (C + N * ( BLOCK_SIZE * by + ty * 4 + 1) + BLOCK_SIZE * bx + tx * 4 )[0] = Csub[1];
//     reinterpret_cast<float4*> (C + N * ( BLOCK_SIZE * by + ty * 4 + 2) + BLOCK_SIZE * bx + tx * 4 )[0] = Csub[2];
//     reinterpret_cast<float4*> (C + N * ( BLOCK_SIZE * by + ty * 4 + 3) + BLOCK_SIZE * bx + tx * 4 )[0] = Csub[3];
// }

// cal offset from row col and ld , in row-major matrix, ld is the width of the matrix
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// transfer float4
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])
template <
    const int BLOCK_SIZE_M,  // width of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // height of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // height of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X,  // width of block of C that each thread calculate
    const bool ENABLE_DOUBLE_BUFFER // whether enable double buffering or not
    > 
__global__ void MatrixMulCUDA5( 
    float * __restrict__ A_Val,
    int* __restrict__ A_col_idx,
    int* __restrict__ A_row_ptr,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int K,
    const int N) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // size of thread block
    const int bszx = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int bszy = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = bszy * bszx;

    // thread id
    const int tid = ty * bszx + tx;

    __shared__ float As[BLOCK_SIZE_M][BLOCK_SIZE_K]; // avoid bank conflict
    __shared__ float Bs[BLOCK_SIZE_K][BLOCK_SIZE_N];
    // registers for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X] = {0};
    // registers for A and B
    float frag_a[THREAD_SIZE_Y];
    float frag_b[THREAD_SIZE_X];
    
    // threads needed to load one row of tile
    // / 4 is because float4 is used
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;
    
    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4;
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;
    
    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;
    
    int row_ptr_start = A_row_ptr[by];
    int row_ptr_end = A_row_ptr[by + 1];
    
    for (int row_ptr = row_ptr_start ; row_ptr < row_ptr_end ; row_ptr = row_ptr + 1) {
        int tile_idx = A_col_idx[row_ptr] * BLOCK_SIZE_K;
        float* A = A_Val + BLOCK_SIZE_M * BLOCK_SIZE_K * row_ptr;
        // load A from global memory to shared memory
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
            FETCH_FLOAT4(As[A_TILE_ROW_START + i][A_TILE_COL]) = FETCH_FLOAT4(A[OFFSET(
                    A_TILE_ROW_START + i, // row
                    A_TILE_COL, // col
                    BLOCK_SIZE_K )]);
        }

        // load B from global memory to shared memory
        #pragma unroll
        for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
            FETCH_FLOAT4(Bs[B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                    tile_idx + B_TILE_ROW_START + i, // row
                    B_TILE_COL + BLOCK_SIZE_N * bx, // col
                    K )]);
        }
    
        __syncthreads();

        // compute c
        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE_K; ++ k) {
            // load A from shared memory to register
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                frag_a[thread_y] = As[ty * THREAD_SIZE_Y + thread_y][k];
            }

            // load B from shared memory to register
            #pragma unroll
            for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
                FETCH_FLOAT4(frag_b[thread_x]) = FETCH_FLOAT4(Bs[k][THREAD_SIZE_X * tx + thread_x]);
            }
            
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                #pragma unroll
                for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                    accum[thread_y][thread_x] += frag_a[thread_y] * frag_b[thread_x];
                }
            }
            
        }
        __syncthreads();
    }

    // store back to C
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
            C[OFFSET(
                BLOCK_SIZE_M * by + ty * THREAD_SIZE_Y + thread_y,
                BLOCK_SIZE_N * bx + tx * THREAD_SIZE_X + thread_x,
                N)] = accum[thread_y][thread_x];
        }
    }
}

int main(int argc, char** argv) {
    if (argc != 5) {
        printf("usage: ./main [M] [K] [N] [Sparsity]\n");
        exit(0);
    }
    size_t M = atoi(argv[1]);
    size_t K = atoi(argv[2]);
    size_t N = atoi(argv[3]);
    size_t Sparsity = atoi(argv[4]);

    size_t bytes = sizeof(float) * M * K;
    float* h_A = (float*)malloc(bytes);
    float* h_B = (float*)malloc(bytes);
    float* h_C = (float*)malloc(bytes);
    float* h_C1 = (float*)malloc(bytes);

    float* d_A;
    float* d_B;
    float* d_C;

    checkCudaErrors(hipMalloc(&d_A, bytes));
    checkCudaErrors(hipMalloc(&d_B, bytes));
    checkCudaErrors(hipMalloc(&d_C, bytes));
    double msecPerMatrixMul[2] = {0, 0};
    double gigaFlops[2] = {0, 0};
    double flopsPerMatrixMul = 2.0 * M * N * K;

    const int BLOCK_SIZE_M = 32;
    const int BLOCK_SIZE_K = 32;
    const int BLOCK_SIZE_N = 32;
    const int THREAD_SIZE_X = 4;
    const int THREAD_SIZE_Y = 4;
    const bool ENABLE_DOUBLE_BUFFER = false;

    int m_block = M / BLOCK_SIZE_M;
    int k_block = K / BLOCK_SIZE_K;
    int nnz_block = m_block * k_block * (Sparsity / 100.0);
    int stride = m_block * k_block / nnz_block;
    float alpha = 1.0;
    float beta = 0;
    
    // 生成A的数据
    for( int i = 0; i < M * K; i++ ) {
        int row = (i / K);
        int col = (i % K);
        int row_block = row / BLOCK_SIZE_M;
        int col_block = col / BLOCK_SIZE_K;
        if ((row_block * k_block + col_block) % stride == 0) h_A[i] = 1;
        else {
            h_A[i] = 0;
        }
    }

    // 生成B的数据
    for( int i = 0; i < K * N; i++ ) {
        if ( i >= K * N / 2) h_B[i] = 2;
        else {
            h_B[i] = 0;
        }
    }
    
    checkCudaErrors(hipMemcpy( d_A, h_A, bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_B, h_B, bytes, hipMemcpyHostToDevice));
    
    
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float msecTotal = 0;
    int nIter = 100;

    // bcsr
    // convert to bcsr mat
    bcsr bcsr_mat{(int)M, (int)K, BLOCK_SIZE_M, BLOCK_SIZE_K};
    cal_block(&bcsr_mat, h_A);

    bcsr_mat.row_ptr = (int*)malloc(sizeof(int) * ( bcsr_mat.m_block + 1 ));
    bcsr_mat.col_idx = (int*)malloc(sizeof(int) * bcsr_mat.nnz_block_num );
    bcsr_mat.val = (float*)malloc(sizeof(float) * bcsr_mat.nnz_block_num * bcsr_mat.m_block_sz * bcsr_mat.n_block_sz);
    
    generate_bcsr(&bcsr_mat, h_A);


    float* val;
    int* col_idx;
    int* row_ptr;

    checkCudaErrors(hipMalloc(&val, sizeof(float) * bcsr_mat.nnz_block_num * bcsr_mat.m_block_sz * bcsr_mat.n_block_sz));
    checkCudaErrors(hipMalloc(&col_idx, sizeof(int) * bcsr_mat.nnz_block_num));
    checkCudaErrors(hipMalloc(&row_ptr, sizeof(int) * ( bcsr_mat.m_block + 1 )));
    
    checkCudaErrors(hipMemcpy( val, bcsr_mat.val, sizeof(float) * bcsr_mat.nnz_block_num * bcsr_mat.m_block_sz * bcsr_mat.n_block_sz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( col_idx, bcsr_mat.col_idx, sizeof(int) * bcsr_mat.nnz_block_num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( row_ptr, bcsr_mat.row_ptr, sizeof(int) * ( bcsr_mat.m_block + 1), hipMemcpyHostToDevice));
    
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
        dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
        MatrixMulCUDA5<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X, ENABLE_DOUBLE_BUFFER>
        <<< dimGrid, dimBlock >>>(val, col_idx, row_ptr, d_B, d_C, K, N);
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy( h_C, d_C, bytes, hipMemcpyDeviceToHost));

    msecPerMatrixMul[0] = msecTotal / nIter;
    gigaFlops[0] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf( "My sparse block gemm Performance= %.0f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[0],
        msecPerMatrixMul[0],
        flopsPerMatrixMul);

    // cusparse csr
    csr csr_mat{(int)M, (int)K}; 
    cal_nnz(&csr_mat, h_A);

    csr_mat.row_ptr = (int*)malloc(sizeof(int) * ( csr_mat.m + 1 ));
    csr_mat.col_idx = (int*)malloc(sizeof(int) * csr_mat.nnz_num );
    csr_mat.val = (float*)malloc(sizeof(float) * csr_mat.nnz_num );

    generate_csr(&csr_mat, h_A);
    
    float* csr_val;
    int* csr_col_idx;
    int* csr_row_ptr;

    checkCudaErrors(hipMalloc(&csr_val, sizeof(float) * csr_mat.nnz_num));
    checkCudaErrors(hipMalloc(&csr_col_idx, sizeof(int) * csr_mat.nnz_num));
    checkCudaErrors(hipMalloc(&csr_row_ptr, sizeof(int) * ( csr_mat.m + 1 )));
    
    checkCudaErrors(hipMemcpy( csr_val, csr_mat.val, sizeof(float) * csr_mat.nnz_num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( csr_col_idx, csr_mat.col_idx, sizeof(int) * csr_mat.nnz_num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( csr_row_ptr, csr_mat.row_ptr, sizeof(int) * ( csr_mat.m + 1 ), hipMemcpyHostToDevice));

    hipsparseHandle_t cusparse_handle;
    hipsparseSpMatDescr_t descrA;
    hipsparseDnMatDescr_t descrB, descrC;

    checkCuSparseErrors(hipsparseCreate(&cusparse_handle));

    checkCuSparseErrors(
        hipsparseCreateDnMat(
            &descrB,
            K,
            M,
            K,
            d_B,
            HIP_R_32F,
            HIPSPARSE_ORDER_COL));
    
    checkCuSparseErrors(
        hipsparseCreateDnMat(
            &descrC,
            M,
            N,
            M,
            d_C,
            HIP_R_32F,
            HIPSPARSE_ORDER_COL));
    
    checkCuSparseErrors (
        hipsparseCreateCsr(&descrA,
            M,
            K,
            csr_mat.nnz_num,
            csr_row_ptr,
            csr_col_idx,
            csr_val,
            HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO,
            HIP_R_32F));

    checkCudaErrors(hipMemcpy( d_B, h_B, bytes, hipMemcpyHostToDevice));
    
    size_t buffer_size;
    checkCuSparseErrors(
        hipsparseSpMM_bufferSize(
                cusparse_handle,
                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                HIPSPARSE_OPERATION_TRANSPOSE,
                &alpha,
                descrA,
                descrB,
                &beta,
                descrC,
                HIP_R_32F,
                HIPSPARSE_CSRMM_ALG1,
                &buffer_size
            ));
    float* externalBuffer;
    checkCudaErrors(hipMalloc(&externalBuffer, buffer_size));
    
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        checkCuSparseErrors(
            hipsparseSpMM(cusparse_handle,
                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                HIPSPARSE_OPERATION_TRANSPOSE,
                &alpha,
                descrA,
                descrB,
                &beta,
                descrC,
                HIP_R_32F,
                HIPSPARSE_CSRMM_ALG1,
                externalBuffer
                )
            );
        
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy( h_C1, d_C, bytes, hipMemcpyDeviceToHost));

    msecPerMatrixMul[1] = msecTotal / nIter;
    gigaFlops[1] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
    printf( "CuSparse Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[1],
        msecPerMatrixMul[1],
        flopsPerMatrixMul);
    
    double eps = 1.e-6;  // machine zero
    bool correct = true;
    for (int i = 0; i < M * N; i++) {
        // h_C1 是转置
        int row = i / N;
        int col = i % N;
        double abs_err = fabs(h_C[i] - h_C1[col * M + row]);
        double dot_length = M;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                    i, h_C[i], h_C1[col * M + row], eps);
            correct = false;
            break;
        }
    }



    printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
    printf("ratio= %f\n", gigaFlops[0] / gigaFlops[1]);
    

    // sort four methods
    // int idx[4] = {0, 1, 2, 3};
    // for ( int i = 0 ; i < 4 ; i ++) {
    //     for ( int j = i + 1 ; j < 4 ; j ++ ) {
    //         if (msecPerMatrixMul[j] <= msecPerMatrixMul[i]) {
    //             int temp_idx = idx[i];
    //             idx[i] = idx[j];
    //             idx[j] = temp_idx;
                
    //             float temp = msecPerMatrixMul[j];
    //             msecPerMatrixMul[j] = msecPerMatrixMul[i];
    //             msecPerMatrixMul[i] = temp;
    //         }
    //     }
    // }

    // printf("\u001b[31m\n");
    // for ( int i = 0 ; i < 4 ; i ++ ) {
    //     if (idx[i] == 0 ) printf("my gemm: %.3f msec\n", msecPerMatrixMul[i]);
    //     else if (idx[i] == 1 ) printf("cublas: %.3f msec\n", msecPerMatrixMul[i]);
    //     else if (idx[i] == 2 ) printf("my block sparse: %.3f msec\n", msecPerMatrixMul[i]);
    //     else if (idx[i] == 3 ) printf("cusparse(csr): %.3f msec\n", msecPerMatrixMul[i]);
    //     else {

    //     }
    // }
    // printf("\u001b[0m\n");
    // Free Memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C1);
}